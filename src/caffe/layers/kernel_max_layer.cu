#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>
#include <cfloat>

#include "caffe/layers/kernel_max_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void KernelForward(const int nthreads, const Dtype* const in,
    const int num, const int channels, const int height, const int width, Dtype* const out) {

  CUDA_KERNEL_LOOP(index, nthreads) {
    // find out the local index
    // find out the local offset
    const int w = index % width;
    const int h = (index / width) % height;
    const int n = index / width / height;
    //const int offset = (n * channels + c) * h * w;
    //const Dtype* const in_slice = in + offset;
    Dtype km=-FLT_MAX;
    for (int c=0; c < channels; ++c) {
        if(in[(n*channels+c)*height*width+h*width+w] > km){
           km=in[(n*channels+c)*height*width+h*width+w];
           //kmid=c;
        }
    }
    out[index] = km;
  }
}

template <typename Dtype>
void KernelMaxLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();//derivative
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int count = top[0]->count();
  channels_ = bottom[0]->channels();
  height_ = bottom[0]->height();
  width_ = bottom[0]->width();
    KernelForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, bottom_data, bottom[0]->num(), channels_,
        height_, width_, top_data);
  CUDA_POST_KERNEL_CHECK;
}

template <typename Dtype>
void KernelMaxLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
}

INSTANTIATE_LAYER_GPU_FUNCS(KernelMaxLayer);
}  // namespace caffe
