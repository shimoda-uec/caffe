#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "caffe/layers/guided_relu_set_layer.hpp"


namespace caffe {

template <typename Dtype>
__global__ void GuidedReLUSetForward(const int n, const Dtype* in,const Dtype* in2,
    Dtype* out, int i) {
  CUDA_KERNEL_LOOP(index, n) {
       for(int e=0;e<i;++e){
         out[e*n+index] = in[e*n+index]*(in2[0*n+index] > 0)*(in[e*n+index]>0);
        }
  }
}
template <typename Dtype>
void GuidedReLUSetLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();//derivative
  const Dtype* bottom_data2 = bottom[1]->gpu_data();//conv value
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int inner_num=bottom[1]->channels()*bottom[1]->height()*bottom[1]->width();
  GuidedReLUSetForward<Dtype><<<CAFFE_GET_BLOCKS(inner_num), CAFFE_CUDA_NUM_THREADS>>>(
               inner_num, bottom_data,bottom_data2,top_data,bottom[0]->num());
            CUDA_POST_KERNEL_CHECK;
}

template <typename Dtype>
void GuidedReLUSetLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
}

INSTANTIATE_LAYER_GPU_FUNCS(GuidedReLUSetLayer);
}  // namespace caffe
