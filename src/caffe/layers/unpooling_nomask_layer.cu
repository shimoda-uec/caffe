#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>


#include "caffe/util/math_functions.hpp"
#include "caffe/layers/unpooling_nomask_layer.hpp"

namespace caffe {

template <typename Dtype>
__global__ void MaxPoolBackward(const int nthreads, const Dtype* const in,
    const int num,
    const int channels, const int height, const int width,
    const int pooled_height, const int pooled_width, const int kernel_h,
    const int kernel_w, const int stride_h, const int stride_w, const int pad_h,
    const int pad_w, Dtype* const out) {

  CUDA_KERNEL_LOOP(index, nthreads) {
    // find out the local index
    // find out the local offset
    const int w = index % pooled_width;
    const int h = (index / pooled_width) % pooled_height;
    const int c = (index / pooled_width / pooled_height) % channels;
    const int n = index / pooled_width / pooled_height / channels;
    const int phstart =
         (h + pad_h < kernel_h) ? 0 : (h + pad_h - kernel_h) / stride_h + 1;
    const int phend = min((h + pad_h) / stride_h + 1, height);
    const int pwstart =
         (w + pad_w < kernel_w) ? 0 : (w + pad_w - kernel_w) / stride_w + 1;
    const int pwend = min((w + pad_w) / stride_w + 1, width);
    Dtype gradient = 0;
    const int offset = (n * channels + c) * height * width;
    const Dtype* const in_slice = in + offset;
      for (int ph = phstart; ph < phend; ++ph) {
        for (int pw = pwstart; pw < pwend; ++pw) {
            gradient += in_slice[ph * width + pw];
        }
      }
    out[index] = gradient;
  }
}




template <typename Dtype>
void UnpoolingNomaskLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
  const int count = top[0]->count();
  const Dtype *bottom_data = bottom[0]->gpu_data();
  Dtype *top_data = top[0]->mutable_gpu_data();
  switch (this->layer_param_.pooling_param().pool()) {
  case PoolingParameter_PoolMethod_MAX:
    MaxPoolBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, bottom_data, bottom[0]->num(), channels_,
        height_, width_, pooled_height_, pooled_width_,
        kernel_h_, kernel_w_, stride_h_, stride_w_, pad_h_, pad_w_,
        top_data);
      break;
  case PoolingParameter_PoolMethod_AVE:
      NOT_IMPLEMENTED;
      break;
  case PoolingParameter_PoolMethod_STOCHASTIC:
      NOT_IMPLEMENTED;
      break;
  }
  CUDA_POST_KERNEL_CHECK;
}

template <typename Dtype>
void UnpoolingNomaskLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
}

INSTANTIATE_LAYER_GPU_FUNCS(UnpoolingNomaskLayer);
} //namespace caffe
